#include "hip/hip_runtime.h"
﻿#define MAX_PASS_SIZE 30
#define MAX_PASS_COUNT 10000000
#define MAX_HASH_COUNT 100
#define INIT_VECTOR_A 0x67452301
#define INIT_VECTOR_B 0xefcdab89
#define INIT_VECTOR_C 0x98badcfe
#define INIT_VECTOR_D 0x10325476
#define MD5_HASH_SIZE 16
#define NOT_FOUND 999999999
#define MAX_PATH_TO_FILE 256

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

typedef unsigned char byte;
typedef unsigned int byte4;

const unsigned long int s[64] = {
	7, 12, 17, 22,   7, 12, 17, 22,   7, 12, 17, 22,  7, 12, 17, 22,
	5,  9, 14, 20,   5,  9, 14, 20,   5,  9, 14, 20,  5,  9, 14, 20,
	4, 11, 16, 23,   4, 11, 16, 23,   4, 11, 16, 23,  4, 11, 16, 23,
	6, 10, 15, 21,   6, 10, 15, 21,   6, 10, 15, 21,  6, 10, 15, 21
};

__device__ unsigned long int dev_s[64] = {
	7, 12, 17, 22,   7, 12, 17, 22,   7, 12, 17, 22,  7, 12, 17, 22,
	5,  9, 14, 20,   5,  9, 14, 20,   5,  9, 14, 20,  5,  9, 14, 20,
	4, 11, 16, 23,   4, 11, 16, 23,   4, 11, 16, 23,  4, 11, 16, 23,
	6, 10, 15, 21,   6, 10, 15, 21,   6, 10, 15, 21,  6, 10, 15, 21
};

const unsigned long int K[64] = {
	0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee,
	0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
	0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
	0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
	0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa,
	0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
	0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed,
	0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
	0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
	0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
	0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05,
	0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
	0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039,
	0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
	0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
	0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391
};

__device__ unsigned long int dev_K[64] = {
	0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee,
	0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
	0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
	0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
	0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa,
	0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
	0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed,
	0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
	0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
	0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
	0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05,
	0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
	0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039,
	0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
	0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
	0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391
};

/* Первый аргумент - строка, хеш которой надо получить.
   Второй аргумент - результат хеширования (пока будет так). */
void md5_get_hash(unsigned char* buff_in, unsigned char* buff_out);
/* MD5 + salt. */
void md5_get_hash_salt(unsigned char* buff_in, unsigned char* salt, unsigned char* buff_out);
void md5_print(unsigned char* hash);
__device__ void md5_get_hash_cuda(unsigned char* buff_in, unsigned char* buff_out);

hipError_t brutWithCuda(unsigned char* passList, unsigned char* hashList, unsigned int* passListSize, unsigned int* hashListSize, unsigned int* result);
void getHashListConfig(unsigned char* hashListOut, unsigned int* hashListSizeOut, FILE* file);
void getPassList(unsigned char* passListOut, unsigned int* passListSizeOut, FILE* file);

__device__ int memcmp_cuda(unsigned char* s1, unsigned char* s2, unsigned int size) {
	unsigned char* s1cpy = s1;
	unsigned char* s2cpy = s2;

	for (; size > 1 && *s1cpy == *s2cpy; ++s1cpy, ++s2cpy, size--);

	if (*s1cpy == *s2cpy) {
		return 0;
	}
	else if (*s1cpy < *s2cpy) return -1;
	else return 1;
}

__global__ void brutKernel(unsigned char* passList, unsigned char* hashList, unsigned int* passListSize, unsigned int* hasListSize, unsigned int* result)
{
	unsigned int bid = blockIdx.x;
	unsigned int tid = threadIdx.x;
	unsigned int i = bid * blockDim.x + tid;
    unsigned char md5_hash[MD5_HASH_SIZE];
    
    if (i < 0 || i >= *passListSize) {
        return;
    }
	
    md5_get_hash_cuda(passList + i * MAX_PASS_SIZE, md5_hash);
    for (unsigned int j = 0; j < *hasListSize; j++) {
        if (memcmp_cuda(hashList + j * MD5_HASH_SIZE, md5_hash, MD5_HASH_SIZE) == 0) {
			result[j] = i;
            break;
        }
    }
} 

int main()
{
	hipError_t cudaStatus;
    FILE* filePassList = NULL;
    FILE* fileConfig = NULL;
	unsigned char* passList = NULL;
	unsigned char* hashList = NULL;
    unsigned int passListSize = 0;
    unsigned int hashListSize = 0;
	unsigned int result[MAX_HASH_COUNT]; for (int i = 0; i < MAX_HASH_COUNT; i++) {result[i] = NOT_FOUND;}
	unsigned char input[MAX_PATH_TO_FILE];
	time_t before = 0, after = 0;

	printf("Enter path to seclist file (type 'def' to use default file): ");
	scanf_s("%s", input, MAX_PATH_TO_FILE);
	if (strcmp((char*)input, "def") == 0) {
		strcpy((char*)input, "10000000.txt");
	}

	fopen_s(&filePassList, (char*)input, "r");
	if (filePassList == NULL) {
		printf("fopen fail 1000000.txt\n");
		return 1;
	}
	fopen_s(&fileConfig, "Config.txt", "r");
	if (fileConfig == NULL) {
		printf("fopen fail Config.txt\n");
		return 1;
	}

	// Выделение памяти под массивы
    // Список паролей из секлиста
	passList = (unsigned char*)calloc(MAX_PASS_COUNT, MAX_PASS_SIZE * sizeof(unsigned char));
	if (passList == NULL) {
		exit(-43);
	}

	// Массив под хеши паролей, которые идут в конфигурационном файле
	hashList = (unsigned char*)calloc(MAX_HASH_COUNT, MD5_HASH_SIZE * sizeof(unsigned char));
	if (hashList == NULL) {
		exit(-43);
	}  

	// Получение списков паролей из файлов
	printf("Getting Hashes from Config file... ");
	getHashListConfig(hashList, &hashListSize, fileConfig);
	printf("OK\n");
	printf("Getting Passwords List from %s pass seclist file... ", input);
	getPassList(passList, &passListSize, filePassList);
	printf("OK\n");

    fclose(filePassList);
	fclose(fileConfig);

	// Brutforce md5 hash of passwords in parallel
	printf("Starting Brutforce with CUDA\n\n");
	time(&before);
	cudaStatus = brutWithCuda(passList, hashList, &passListSize, &hashListSize, result);
	time(&after);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	printf("Finished!\n\n");

	for (unsigned int i = 0; i < hashListSize; i++) {
		printf("Hash: ");
		md5_print(hashList + i * MD5_HASH_SIZE);

		printf(" Password: ");
		if (result[i] != NOT_FOUND) {
			printf("%s\n", passList + result[i] * MAX_PASS_SIZE);
		}
		else {
			printf("not found\n");
		}
	}

	printf("\nTime: %f sec.\n", difftime(after, before));

	free(passList);
	free(hashList);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

 //Helper function for using CUDA.
hipError_t brutWithCuda(unsigned char* passList, unsigned char* hashList, unsigned int* passListSize, unsigned int* hashListSize, unsigned int* result)
{
	unsigned char* dev_passList = 0;
	unsigned char* dev_hashList = 0;
	unsigned int* dev_passListSize = 0;
	unsigned int* dev_hasListsize = 0;
	unsigned int* dev_result = 0;

    hipError_t cudaStatus;
	int n_blocks = (MAX_PASS_COUNT + 255) / 256;
	int threads_per_block = 256;
	dim3 grid(n_blocks, 1, 1);
	dim3 threads(threads_per_block, 1, 1);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)  
	// Память под результат
	cudaStatus = hipMalloc((void**)&dev_result, MAX_HASH_COUNT * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_result, result, MAX_HASH_COUNT * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	// Память под секлист
    cudaStatus = hipMalloc((void**)&dev_passList, MAX_PASS_COUNT * MAX_PASS_SIZE * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(dev_passList, passList, MAX_PASS_COUNT * MAX_PASS_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Память под хэши из конфига
	cudaStatus = hipMalloc((void**)&dev_hashList, MAX_HASH_COUNT * MD5_HASH_SIZE * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_hashList, hashList, MAX_HASH_COUNT * MD5_HASH_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Память под число хэшей
	cudaStatus = hipMalloc((void**)&dev_hasListsize, sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_hasListsize, hashListSize, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Память под число паролей из секлиста
	cudaStatus = hipMalloc((void**)&dev_passListSize, sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_passListSize, passListSize, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	brutKernel <<<grid, threads >>> (dev_passList, dev_hashList, dev_passListSize, dev_hasListsize, dev_result);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, dev_result, sizeof(unsigned int) * MAX_HASH_COUNT, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_passList);
	hipFree(dev_hashList);
    hipFree(dev_hasListsize);
    hipFree(dev_passListSize);
    hipFree(dev_result);
    
    return cudaStatus;
}

// Get max 100 pass from Config file
void getHashListConfig(unsigned char* hashListOut, unsigned int* hashListSizeOut, FILE* file) {
    unsigned char pass[MAX_PASS_SIZE];
    unsigned char salt[MAX_PASS_SIZE];
	unsigned char c;

    while (!feof(file))
    {
		c = '\0';
		fscanf(file, "%s", pass); fscanf(file, "%c", &c);
		if (c == ' ') {
			fscanf(file, "%s", salt);
		}
		else
		{
			salt[0] = '\0';
		}

        if (pass[0] != '\0' && pass[0] != '\n' && *hashListSizeOut < 100) {
            md5_get_hash_salt(pass, salt, hashListOut + (*hashListSizeOut)++ * MD5_HASH_SIZE);
        }
    }
}

void getPassList(unsigned char* passListOut, unsigned int* passListSizeOut, FILE* file) {
    for (int i = 0; (!feof(file)); i++) {
        (*passListSizeOut)++;
        fscanf_s(file, "%s", passListOut + i * MAX_PASS_SIZE, 30);
    }
}

/* Преобразование из Big-end в Little-end и наоборот. */
inline unsigned int bswap(unsigned int v)
{
	return (v >> 24) | ((v >> 8) & 0xff00)
		| ((v << 8) & 0xff0000) | (v << 24);
}

inline byte4 rol(byte4 a, int offset)
{
	return a << offset | a >> (32 - offset);
}

__device__ inline byte4 rol_cuda(byte4 a, int offset)
{
	return a << offset | a >> (32 - offset);
}

__device__ inline unsigned int bswap_cuda(unsigned int v)
{
	return (v >> 24) | ((v >> 8) & 0xff00)
		| ((v << 8) & 0xff0000) | (v << 24);
}

void md5_get_hash(unsigned char* buff_in, unsigned char* buff_out) {
	byte4	A0 = INIT_VECTOR_A,
		B0 = INIT_VECTOR_B,
		C0 = INIT_VECTOR_C,
		D0 = INIT_VECTOR_D;
	unsigned long int len_bytes = strlen((char*)buff_in),     // Длина оригинальной входной строки в байтах
		len_bits = len_bytes * 8,        // Длина оригинальной входной строки в битах
		new_len_bits = 0,		           // Длина новой строки, после подготовки в битах
		new_len_bytes = 0;			   // Длина новой строки, после подготовки в байтах
	int N;										       // N из формулы 512 * N + 448
	byte* byte_buff = NULL;						       // Массив байт, содержащий оригинальную строку + выравшивающие биты + 64 бита длины оригинальной строки

	/* 1. Подготовка массива байт. */
	// 1.a. Расчет новой длины после добовления выравнивающих бит по формуле: 512 * N + 448
	N = (((int)len_bits) + 1 - 448) / 512; // + 1 означает, что в конец обязательно добавлен 1 бит (из алгоритма).
	new_len_bits = 512 * N + 448;

	// 1.b. Добавить 64 бита (из алгоритма).
	new_len_bits += 64;

	// 1.c. Выделить и заполнить главный буффер.
	new_len_bytes = new_len_bits / 8;

	byte_buff = (byte*)calloc(new_len_bytes, sizeof(byte));
	if (byte_buff == NULL) {
		printf("calloc fail! (md5)\n");
		return;
	}

	memset(byte_buff, 0x00, sizeof(byte) * new_len_bytes); // Инициализация всеми нулями (тогда не нужно будет вручную ставить нулевые биты в конце
	memcpy(byte_buff, buff_in, sizeof(byte) * len_bytes); // Копирование исходной строки
	byte_buff[len_bytes] = 0x80; // Добавить 1 бит к исходной строке (1000 0000)
	//len_bits = bswap(len_bits); // Тут должно быть преобразование из Big-endian в Little-endian, но оно уже преобразовано, то ли сам виндовс little-endian, то ли memcpy копирует в обратном порядке.
	memcpy(byte_buff + new_len_bytes - 8, &len_bits, sizeof(byte) * 4); // Добавить в конец длину исходной строки в формате 64 бит (little-endian).

	//// ОтладОЧКА
	//for (unsigned long int i = 0; i < new_len_bytes; i++) {
	//	printf("%x ", byte_buff[i]);
	//}
	//printf("\n");

	/* 2. Хеширование. */
	for (unsigned int chunk = 0; chunk < new_len_bytes; chunk += 64) { // Разбиваем строку на блоки по 512 бит = 64 байт
		byte4	A = A0,
			B = B0,
			C = C0,
			D = D0;
		byte4* block = (byte4*)(byte_buff + chunk); // 32-х битный блок (4 байт) из 512 (64 байт) битного чанка.
		for (int i = 0; i < 64; i++) {
			byte4 F;
			unsigned int g; // Номер 32-х битного блока (4 байт) из 512 (64 байт) битного чанка.

			if (0 <= i && i <= 15) {
				F = (B & C) | ((~B) & D);	// Функция F
				g = i;
			}
			else if (16 <= i && i <= 31) {
				F = (D & B) | ((~D) & C);	// Функция G
				g = (5 * i + 1) % 16;
			}
			else if (32 <= i && i <= 47) {
				F = B ^ C ^ D;				// Функция H
				g = (3 * i + 5) % 16;
			}
			else { // (48 <= i && i <= 63)
				F = C ^ (B | (~D));			// Функция I
				g = (7 * i) % 16;
			}

			F = F + A + K[i] + block[g];
			A = D;
			D = C;
			C = B;
			byte4 test = rol(F, s[i]);
			B = B + rol(F, s[i]);
		}

		A0 += A;
		B0 += B;
		C0 += C;
		D0 += D;
	}

	/* Запись результата. */
	/*A0 = bswap(A0);
	B0 = bswap(B0);
	C0 = bswap(C0);
	D0 = bswap(D0);*/
	memcpy(buff_out, &A0, sizeof(byte4));
	memcpy(buff_out + 4, &B0, sizeof(byte4));
	memcpy(buff_out + 8, &C0, sizeof(byte4));
	memcpy(buff_out + 12, &D0, sizeof(byte4));

	free(byte_buff);
}

void md5_get_hash_salt(unsigned char* buff_in, unsigned char* salt, unsigned char* buff_out) {
	unsigned char* new_buff = NULL;
	size_t in_len = strlen((char*)buff_in), salt_len = strlen((char*)salt);

	new_buff = (unsigned char*)malloc(in_len + salt_len + 1);
	if (new_buff == NULL) {
		printf("malloc fail! (salt)\n");
		return;
	}

	memcpy(new_buff, buff_in, in_len);
	memcpy(new_buff + in_len, salt, salt_len + 1);

	md5_get_hash(new_buff, buff_out);

	free(new_buff);
}

__device__ int strlen_cuda(unsigned char* s) {
	unsigned char* scpy = s;
	int size = 0;

	for (; *scpy != '\0'; size++, scpy++);

	return size;
}

__device__ void md5_get_hash_cuda(unsigned char* buff_in, unsigned char* buff_out) {
	byte4	A0 = INIT_VECTOR_A,
		B0 = INIT_VECTOR_B,
		C0 = INIT_VECTOR_C,
		D0 = INIT_VECTOR_D;
	unsigned long int len_bytes = strlen_cuda(buff_in),     // Длина оригинальной входной строки в байтах
		len_bits = len_bytes * 8,        // Длина оригинальной входной строки в битах
		new_len_bits = 0,		           // Длина новой строки, после подготовки в битах
		new_len_bytes = 0;			   // Длина новой строки, после подготовки в байтах
	int N;										       // N из формулы 512 * N + 448
	byte* byte_buff = NULL;						       // Массив байт, содержащий оригинальную строку + выравшивающие биты + 64 бита длины оригинальной строки

	/* 1. Подготовка массива байт. */
	// 1.a. Расчет новой длины после добовления выравнивающих бит по формуле: 512 * N + 448
	N = (((int)len_bits) + 1 - 448) / 512; // + 1 означает, что в конец обязательно добавлен 1 бит (из алгоритма).
	new_len_bits = 512 * N + 448;

	// 1.b. Добавить 64 бита (из алгоритма).
	new_len_bits += 64;

	// 1.c. Выделить и заполнить главный буффер.
	new_len_bytes = new_len_bits / 8;

	byte_buff = (byte*)malloc(sizeof(byte) * new_len_bytes);
	if (byte_buff == NULL) {
		printf("calloc fail! (md5)\n");
		return;
	}

	memset(byte_buff, 0x00, sizeof(byte) * new_len_bytes); // Инициализация всеми нулями (тогда не нужно будет вручную ставить нулевые биты в конце
	memcpy(byte_buff, buff_in, sizeof(byte) * len_bytes); // Копирование исходной строки
	byte_buff[len_bytes] = 0x80; // Добавить 1 бит к исходной строке (1000 0000)
	//len_bits = bswap(len_bits); // Тут должно быть преобразование из Big-endian в Little-endian, но оно уже преобразовано, то ли сам виндовс little-endian, то ли memcpy копирует в обратном порядке.
	memcpy(byte_buff + new_len_bytes - 8, &len_bits, sizeof(byte) * 4); // Добавить в конец длину исходной строки в формате 64 бит (little-endian).

	//// ОтладОЧКА
	//for (unsigned long int i = 0; i < new_len_bytes; i++) {
	//	printf("%x ", byte_buff[i]);
	//}
	//printf("\n");

	/* 2. Хеширование. */
	for (unsigned int chunk = 0; chunk < new_len_bytes; chunk += 64) { // Разбиваем строку на блоки по 512 бит = 64 байт
		byte4	A = A0,
			B = B0,
			C = C0,
			D = D0;
		byte4* block = (byte4*)(byte_buff + chunk); // 32-х битный блок (4 байт) из 512 (64 байт) битного чанка.
		for (int i = 0; i < 64; i++) {
			byte4 F;
			unsigned int g; // Номер 32-х битного блока (4 байт) из 512 (64 байт) битного чанка.

			if (0 <= i && i <= 15) {
				F = (B & C) | ((~B) & D);	// Функция F
				g = i;
			}
			else if (16 <= i && i <= 31) {
				F = (D & B) | ((~D) & C);	// Функция G
				g = (5 * i + 1) % 16;
			}
			else if (32 <= i && i <= 47) {
				F = B ^ C ^ D;				// Функция H
				g = (3 * i + 5) % 16;
			}
			else { // (48 <= i && i <= 63)
				F = C ^ (B | (~D));			// Функция I
				g = (7 * i) % 16;
			}

			F = F + A + dev_K[i] + block[g];
			A = D;
			D = C;
			C = B;
			B = B + rol_cuda(F, dev_s[i]);
		}

		A0 += A;
		B0 += B;
		C0 += C;
		D0 += D;
	}

	/* Запись результата. */
	/*A0 = bswap(A0);
	B0 = bswap(B0);
	C0 = bswap(C0);
	D0 = bswap(D0);*/
	memcpy(buff_out, &A0, sizeof(byte4));
	memcpy(buff_out + 4, &B0, sizeof(byte4));
	memcpy(buff_out + 8, &C0, sizeof(byte4));
	memcpy(buff_out + 12, &D0, sizeof(byte4));

	free(byte_buff);
}

void md5_print(unsigned char* hash) {
	for (int i = 0; i < MD5_HASH_SIZE; i++) {
		printf("%02x", hash[i]);
	}
}